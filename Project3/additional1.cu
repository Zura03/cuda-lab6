#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<hip/hip_runtime.h>
#include<>


__global__ void kernel(char* da, char* db, char* dlen) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx < *dlen) {
		int si = *dlen - idx - 1;
		db[si] = da[idx];
	}
}
int main() {
	int n;
	printf("Enter N: ");
	scanf("%d", &n);

	char* a, * b;
	char* da, * db, * dlen;
	int size = n * sizeof(char);
	a = (char*)malloc(size);
	b = (char*)malloc(size);
	memset(b, 0, size);

	printf("Enter string A: ");
	scanf("%s", a);
	int len = strlen(a);

	hipMalloc((void**)&da, size);
	hipMalloc((void**)&db, size);
	hipMalloc((void**)&dlen, sizeof(int));

	hipMemcpy(da, a, size, hipMemcpyHostToDevice);
	hipMemcpy(db, b, size, hipMemcpyHostToDevice);
	hipMemcpy(dlen, &len, sizeof(int), hipMemcpyHostToDevice);

	kernel << <1, len >> > (da, db, dlen);

	hipMemcpy(b, db, size, hipMemcpyDeviceToHost);

	b[len] = '\0';
	printf("%s", b);

	return 0;
}