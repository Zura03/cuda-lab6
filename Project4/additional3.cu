#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<hip/hip_runtime.h>
#include<>


__global__ void kernel(char* da, char* db, int* dlen) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx < *dlen) {
		int si = 0;
		for (int i = 0; i < idx; i++) {
			//si += (*dlen) - (*dlen - idx);
			si += i + 1;
		}

		int total_chars = idx + 1;
		for (int i = 0; i < total_chars; i++)
			db[si++] = da[idx];
			//db[si + i] = da[idx];
	}
}
int main() {
	int n;
	printf("Enter N: ");
	scanf("%d", &n);

	char* a, * b;
	char* da, * db;
	int* dlen;
	int size = n * sizeof(char);
	a = (char*)malloc(size);

	printf("Enter string A: ");
	scanf("%s", a);
	int len = strlen(a);

	int blen = 0;
	for (int i = 0; i < len; i++)
		blen += i + 1;

	int bsize = blen * sizeof(char);
	b = (char*)malloc(bsize);
	memset(b, 0, bsize);

	hipMalloc((void**)&da, size);
	hipMalloc((void**)&db, bsize);
	hipMalloc((void**)&dlen, sizeof(int));

	hipMemcpy(da, a, size, hipMemcpyHostToDevice);
	hipMemcpy(db, b, bsize, hipMemcpyHostToDevice);
	hipMemcpy(dlen, &len, sizeof(int), hipMemcpyHostToDevice);

	kernel << <1, len >> > (da, db, dlen);

	hipMemcpy(b, db, bsize, hipMemcpyDeviceToHost);

	b[blen] = '\0';
	printf("%s", b);

	return 0;
}