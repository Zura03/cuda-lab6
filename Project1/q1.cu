#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<>
#include<string.h>
#define N 1024

__global__ void CUDA_count(char* a, char* b, int* len, int* wordlen, int* cnt) {
	int idx = threadIdx.x;

	int flag = 1;
	if (idx + *wordlen <= *len) {
		for (int i = 0; i < *wordlen; i++) {
			if (a[idx + i] != b[i]) {
				flag = 0;
				break;
			}
		}
		if (flag == 1)
			atomicAdd(cnt, 1);
	}
}
int main() {
	char a[N], b[N];
	int count = 0, len, wordlen, res;

	char* d_a, * d_b;
	int* d_count, * d_len, * d_wordlen;
	printf("Enter string: ");
	scanf("%s", a);
	printf("Enter word: ");
	scanf("%s", b);

	len = strlen(a);
	wordlen = strlen(b);

	hipMalloc((void**)&d_a, strlen(a) * sizeof(char));
	hipMalloc((void**)&d_b, strlen(b) * sizeof(char));
	hipMalloc((void**)&d_count, sizeof(int));
	hipMalloc((void**)&d_len, sizeof(int));
	hipMalloc((void**)&d_wordlen, sizeof(int));

	hipMemcpy(d_a, a, strlen(a) * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, strlen(b) * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(d_wordlen, &wordlen, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_len, &len, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_count, &count, sizeof(int), hipMemcpyHostToDevice);

	CUDA_count << <1, strlen(a) >> > (d_a, d_b, d_len, d_wordlen, d_count);


	hipMemcpy(&res, d_count, sizeof(int), hipMemcpyDeviceToHost);

	printf("%d", res);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_len);
	hipFree(d_count);
	hipFree(d_wordlen);

	return 0;
}