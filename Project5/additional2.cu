#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<hip/hip_runtime.h>
#include<>

/*
__global__ void kernel(char* da, char* db, int* dlen, int* N) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;

	if (idx < *dlen) {
		int si = idx;

		/*int total_chars = *N - 1;
		for (int i = 0; i < total_chars; i++) {
			si = si + (*dlen);
			db[si] = da[idx];
			//db[si + i] = da[idx];
		}

		for (int i = 1; i < *N; i++) {
			si += *dlen;
			db[si] = da[idx];
		}
	}

}*/

__global__ void kernel(char* da, char* db, int* dlen, int* N) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	//int si = 0;
	if (idx < *dlen) {
		for (int i = 0; i < *N; i++) {
			int si = idx + (*dlen)*i;
			db[si] = da[idx];
		}
	}
}
int main() {
	int n, N;
	printf("Enter string length: ");
	scanf("%d", &n);

	char* a, * b;
	char* da, * db;
	int* dlen, * dN;
	int size = n * sizeof(char);
	a = (char*)malloc(size);

	printf("Enter string A: ");
	scanf("%s", a);
	int len = strlen(a);

	printf("Enter N: ");
	scanf("%d", &N);

	int blen = N * len;

	int bsize = blen * sizeof(char);
	b = (char*)malloc(bsize);
	memset(b, 0, bsize);

	hipMalloc((void**)&da, size);
	hipMalloc((void**)&db, bsize);
	hipMalloc((void**)&dlen, sizeof(int));
	hipMalloc((void**)&dN, sizeof(int));

	hipMemcpy(da, a, size, hipMemcpyHostToDevice);
	hipMemcpy(db, b, bsize, hipMemcpyHostToDevice);
	hipMemcpy(dlen, &len, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dN, &N, sizeof(int), hipMemcpyHostToDevice);

	kernel << <1, len >> > (da, db, dlen, dN);

	hipMemcpy(b, db, bsize, hipMemcpyDeviceToHost);

	b[blen] = '\0';
	printf("%s", b);

	return 0;
}